#include "hip/hip_runtime.h"
#include "floyd_warshall_algo.cuh"

using namespace std;

void _showPath(int start,int end,const vector<Piii> &path,const int *D,const int N){
	cout<<"\nHere is the shortest cost path from "<<start<< " to "<<end<<", at a total cost of "<<D[start*N+end]<<".\n";
	for(int i=path.size()-1;i>=0;--i){
		cout<<"From "<<path[i].first.first<<" to "<<path[i].first.second<<" at a cost of "<<path[i].second<<'\n';
	}
	cout<<'\n';
}

bool _getPath(int curEdge, int nxtEdge,vector<Piii> &path,const int *D, const int *Dpath,const int N){
	int curIdx=curEdge*N+nxtEdge;
	if(D[curIdx]>=INF)return false;
	if(Dpath[curIdx]==-1){//end of backwards retracement
		path.push_back(make_pair(make_pair(curEdge,nxtEdge),D[curIdx]));
		return true;
	}else{//record last edge cost and move backwards
		path.push_back(make_pair(make_pair(Dpath[curIdx],nxtEdge),D[Dpath[curIdx]*N+nxtEdge]));
		return _getPath(curEdge,Dpath[curIdx],path,D,Dpath,N);
	}
}

void _get_full_paths(const int *D, const int *Dpath, const int N, int *roots){
	int start_vertex=-1,end_vertex=-1;
	vector<Piii> path;

	//*
	path.clear();
	//start_vertex = 975;
	//end_vertex = 997;
	start_vertex = 1018;
	end_vertex = 1039;

	if(_getPath(start_vertex, end_vertex,path,D,Dpath,N)){
		_showPath(start_vertex,end_vertex,path,D,N);

	}else{
		cout<<"\nThere does not exist valid a path between "<<start_vertex<<" , and "<<end_vertex<<'\n';
	}

	for(int j=0,i=path.size()-1;i>=0;++j,--i){
			roots[j] = path[i].first.first;
	}
	//*/

	/*
	do{
		path.clear();
		cout<<"Enter start vertex #:";
		cin>>start_vertex;
		cout<<"Enter dest vertex(enter negative number to exit) #:";
		cin>>end_vertex;
		if(start_vertex<0 || start_vertex>=N || end_vertex<0 || end_vertex>=N)return;

		if(_getPath(start_vertex, end_vertex,path,D,Dpath,N)){
			_showPath(start_vertex,end_vertex,path,D,N);

		}else{
			cout<<"\nThere does not exist valid a path between "<<start_vertex<<" , and "<<end_vertex<<'\n';

		}
	}while(1);
	//*/
}

__global__ void _Wake_GPU(int reps){
	int idx=blockIdx.x*blockDim.x + threadIdx.x;
	if(idx>=reps)return;
}

__global__ void _GPU_Floyd_kernel(int k, int *G,int *P, int N){//G will be the adjacency matrix, P will be path matrix
	int col=blockIdx.x*blockDim.x + threadIdx.x;
	if(col>=N)return;
	int idx=N*blockIdx.y+col;

	__shared__ int best;
	if(threadIdx.x==0)
		best=G[N*blockIdx.y+k];
	__syncthreads();
	if(best==INF)return;
	int tmp_b=G[k*N+col];
	if(tmp_b==INF)return;
	int cur=best+tmp_b;
	if(cur<G[idx]){
		G[idx]=cur;
		P[idx]=k;
	}
}

void _GPU_Floyd(int *H_G, int *H_Gpath, const int N){
	//allocate device memory and copy graph data from host
	int *dG,*dP;
	int numBytes=N*N*sizeof(int);
	hipError_t err=hipMalloc((int **)&dG,numBytes);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMalloc((int **)&dP,numBytes);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	//copy from host to device graph info
	err=hipMemcpy(dG,H_G,numBytes,hipMemcpyHostToDevice);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMemcpy(dP,H_Gpath,numBytes,hipMemcpyHostToDevice);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	dim3 dimGrid((N+BLOCK_SIZE-1)/BLOCK_SIZE,N);

	for(int k=0;k<N;k++){//main loop
		_GPU_Floyd_kernel<<<dimGrid,BLOCK_SIZE>>>(k,dG,dP,N);
		err = hipDeviceSynchronize();
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	}
	//copy back memory
	err=hipMemcpy(H_G,dG,numBytes,hipMemcpyDeviceToHost);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMemcpy(H_Gpath,dP,numBytes,hipMemcpyDeviceToHost);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	//free device memory
	err=hipFree(dG);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipFree(dP);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
}

void _generateCustomGraph(int *G, int N){
	FILE *dataFile = fopen("data.txt", "r");
	if(dataFile != NULL){
		cout<<"Successfully opened file.\n";
	} else {
		cout<<"File not found.\n";
	}

	for(int i=0; i < 2016*2016; i++){
		fscanf(dataFile, "%d", &G[i]);
	}

	fclose(dataFile);
}
