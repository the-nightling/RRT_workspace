#include "hip/hip_runtime.h"
////////////////////////////////////////////
// INCLUDES
////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#include "RRT.cuh"


////////////////////////////////////////////
// CUDA KERNELS
////////////////////////////////////////////

/*
 * Initializes CUDA RNG
 */
__global__ void RNG_setup_kernel(hiprandState *state) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;		// thread id
	hiprand_init(1234, idx, 0, &state[idx]);	// using seed 1234 (change to time at a later stage)
}

/*
 * Initializes adjacent matrix
 */
__global__ void init_adj_matrix_kernel(int * adjacency_matrix){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i=0; i < NUM_THREADS*NUM_BLOCKS; i++){
		int index = idx * NUM_THREADS*NUM_BLOCKS + i;
		if(index % (NUM_THREADS*NUM_BLOCKS + 1) == 0){
			adjacency_matrix[index] = 0;
		}else{
			adjacency_matrix[index] = 9999;
			//adjacency_matrix[index] = 0;
		}
	}
}

/*
 * Main kernel; Contains RRT algorithm
 */
__global__ void RRT_kernel(hiprandState *my_curandstate, int *adjacency_matrix,
		double * path_solutions, double * control_solutions, double* tmp) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;		// thread id

	// computing initial state
	double start_state[] = { ANG_POS_MIN, ANG_VEL_MIN}; // initial state; angle position measured from x-axis
	start_state[0] += ((idx % GRID_X) * 2 * DELTA_X) + (2 * DELTA_X);
	start_state[1] += (((idx / GRID_X) % (GRID_Y*NUM_BLOCKS)) * 2 * DELTA_Y) + (2 * DELTA_Y);

	tmp[2*idx] = start_state[0];
	tmp[2*idx+1] = start_state[1];

	// automate goal placement around initial state
	double end_state[NUM_OF_GOAL_STATES][DIMENSIONS] = {{0}};
	int goal_idx;
	for(goal_idx = 0; goal_idx < pow((float)3,(float)DIMENSIONS); goal_idx++)
	{
		if(goal_idx < NUM_OF_GOAL_STATES/2){
			end_state[goal_idx][0] = start_state[0] + ((goal_idx%3) - 1)*2*DELTA_X;
			end_state[goal_idx][1] = start_state[1] + (((goal_idx/3)%3) - 1)*2*DELTA_Y;
		}else if(goal_idx > NUM_OF_GOAL_STATES/2){
			end_state[goal_idx-1][0] = start_state[0] + ((goal_idx%3) - 1)*2*DELTA_X;
			end_state[goal_idx-1][1] = start_state[1] + (((goal_idx/3)%3) - 1)*2*DELTA_Y;
		}

	}

	double state_limits[2][2] = {
			{ start_state[0] - 3 * DELTA_X, start_state[0] + 3 * DELTA_X },
			{ start_state[1] - 3 * DELTA_Y, start_state[1] + 3 * DELTA_Y }
	}; // state limits; angular position between -pi & pi rad; angular velocity between -10 & 10 rad/s

	// control torques to be used: linspace(-5,5,20)
	//*
	double discrete_control_torques[] = { -5.0000, -4.4737, -3.9474, -3.4211, -2.8947, -2.3684,
			-1.8421, -1.3158, -0.7895, -0.2632, 5.0000, 4.4737, 3.9474, 3.4211,
			2.8947, 2.3684, 1.8421, 1.3158, 0.7895, 0.2632 };
	//*/
	/*
	double discrete_control_torques[] = { -1.0000, -0.8947, -0.7895, -0.6842, -0.5789, -0.4737, -0.3684, -0.2632, -0.1579, -0.0526,
			1.0000, 0.8947, 0.7895, 0.6842, 0.5789, 0.4737, 0.3684, 0.2632, 0.1579, 0.0526};
	//*/
	int number_of_discrete_torques = (int) (sizeof(discrete_control_torques) / sizeof(discrete_control_torques[0]));

	double time_step = 0.02; // time interval between application of subsequent control torques

	// static memory allocation
	double random_state[DIMENSIONS];        // stores a state
	double next_state[DIMENSIONS];

	double RRT_tree[NUM_OF_ITERATIONS][DIMENSIONS];	// stores tree
	int x, y;
	for (x = 0; x < NUM_OF_ITERATIONS; x++) {	// initialize tree to initial state
		RRT_tree[x][0] = start_state[0];
		RRT_tree[x][1] = start_state[1];
	}

	//int adjMatrix[NUM_THREADS][NUM_THREADS];
	//memset(adjMatrix, 0, sizeof(int)*NUM_THREADS*NUM_THREADS);

	int parent_state_index[NUM_OF_ITERATIONS]; // stores index of parent state for each state in graph RRT_tree
	int control_action_index[NUM_OF_ITERATIONS]; // stores index of control actions in discrete_control_torques (each state will use a control action value in discrete_control_torques)
	double u_path[NUM_OF_GOAL_STATES][LENGTH_OF_SOLN_PATH]; // stores sequence of control actions (solution to problem)
	double x_path[NUM_OF_GOAL_STATES][LENGTH_OF_SOLN_PATH][DIMENSIONS];
	for (y = 0; y < NUM_OF_GOAL_STATES; y++) {
		for (x = 0; x < LENGTH_OF_SOLN_PATH; x++) {	// initialize tree to initial state
			x_path[y][x][0] = 0;
			x_path[y][x][1] = 0;
			u_path[y][x] = 0;
		}
	}
	int state_index = 0;    // stores sequence of states joining initial to goal state
	double temp_achievable_states[20][DIMENSIONS]; // stores temporary achievable states from a particular vertex; 20 is length of discrete_control_torques

	double distance_square_values[NUM_OF_ITERATIONS];  // stores distance square values

	int goal_index;
	int not_found[NUM_OF_GOAL_STATES] = {0};
	for(int i=0; i < NUM_OF_GOAL_STATES;i++)
		not_found[i] = 1;
	int weight = 0;

	// keep growing RRT until goal found or run out of iterations
	int iteration;
	for (iteration = 1; iteration < NUM_OF_ITERATIONS; iteration++) {
		// get random state
		random_state[0] = hiprand_uniform(my_curandstate + idx) * (state_limits[0][1] - state_limits[0][0]) + state_limits[0][0];
		random_state[1] = hiprand_uniform(my_curandstate + idx) * (state_limits[1][1] - state_limits[1][0]) + state_limits[1][0];

		// find distances between that state point and every vertex in RRT
		euclidianDistSquare(random_state, RRT_tree, iteration, distance_square_values);

		// select RRT vertex closest to the state point
		int nearest_state_index = findMin(distance_square_values, iteration);

		// from the closest RRT vertex, compute all the states that can be reached,
		// given the pendulum dynamics and available torques
		int ui;
		for (ui = 0; ui < number_of_discrete_torques; ui++) {
			pendulumDynamics(RRT_tree[nearest_state_index], discrete_control_torques[ui], next_state);
			temp_achievable_states[ui][0] = RRT_tree[nearest_state_index][0] + time_step * next_state[0];
			temp_achievable_states[ui][1] = RRT_tree[nearest_state_index][1] + time_step * next_state[1];
		}

		// select the closest reachable state point
		euclidianDistSquare(random_state, temp_achievable_states, number_of_discrete_torques, distance_square_values);
		ui = findMin(distance_square_values, number_of_discrete_torques);
		random_state[0] = temp_achievable_states[ui][0];
		random_state[1] = temp_achievable_states[ui][1];

		// if angular position is greater than pi rads, wrap around
		if (random_state[0] > M_PI || random_state[0] < -M_PI)
			random_state[0] = fmod((random_state[0] + M_PI), (2 * M_PI)) - M_PI;

		// link reachable state point to the nearest vertex in the tree
		RRT_tree[iteration][0] = random_state[0];
		RRT_tree[iteration][1] = random_state[1];
		parent_state_index[iteration] = nearest_state_index;
		control_action_index[iteration] = ui;

		// if tree has grown near enough to one of the surrounding goal states
		// set that particular goal state to 'found'
		// save path from initial state to that goal state
		for (goal_index = 0; goal_index < NUM_OF_GOAL_STATES; goal_index++) {
			if (not_found[goal_index] == 1
					&& (random_state[0] <= end_state[goal_index][0] + 0.05)
					&& (random_state[0] >= end_state[goal_index][0] - 0.05)) {
				if ((random_state[1] <= end_state[goal_index][1] + 0.25)
						&& (random_state[1] >= end_state[goal_index][1] - 0.25)) {

					not_found[goal_index] = 0;
					state_index = iteration;
					int length_of_soln = 0;
					while (state_index != 0) {
						u_path[goal_index][length_of_soln] = discrete_control_torques[control_action_index[state_index]];
						x_path[goal_index][length_of_soln][0] = RRT_tree[state_index][0];
						x_path[goal_index][length_of_soln][1] = RRT_tree[state_index][1];
						length_of_soln++;

						state_index = parent_state_index[state_index];
					}
				}
			}
		}
	}


	// Update adjacency matrix:
	// for each goal state surrounding an initial state,
	// if the goal state has been reached,
	// if tree is growing near border of phase space, check if tree is growing within state space limits
	// set respective flag in adjacency matrix to 1 (or to a weight)
	//*
	int offset[8] = {-43,-42,-41,-1,1,41,42,43};
	int offset_idx = 0;
	weight = 1;
	int k;
	for (k = 0; k < NUM_OF_GOAL_STATES; k++) {
		if (not_found[k] == 0) {
			offset_idx = offset[k];
			if((idx * NUM_THREADS * NUM_BLOCKS + idx + offset_idx >= 0) && (idx * NUM_THREADS * NUM_BLOCKS + idx + offset_idx < NUM_RESULTS_PER_THREAD * NUM_THREADS * NUM_BLOCKS)){
				if((end_state[k][0] > ANG_POS_MIN+DELTA_X) && (end_state[k][0] < ANG_POS_MAX-DELTA_X) &&
						(end_state[k][1] > ANG_VEL_MIN+DELTA_Y) && (end_state[k][1] < ANG_VEL_MAX-DELTA_Y) ){
					adjacency_matrix[idx * NUM_THREADS * NUM_BLOCKS + idx + offset_idx] = weight;
				}
			}
		}
	}
	//*/

	//* copy path results of algorithm to device results array
	int i, j;
	int num_of_goals = NUM_OF_GOAL_STATES;
	for (j = 0; j < num_of_goals; j++) {
		for (i = 0; i < LENGTH_OF_SOLN_PATH; i++) {
			path_solutions[idx * DIMENSIONS * num_of_goals * LENGTH_OF_SOLN_PATH + j * DIMENSIONS * LENGTH_OF_SOLN_PATH + DIMENSIONS * i] = x_path[j][i][0];
			path_solutions[idx * DIMENSIONS * num_of_goals * LENGTH_OF_SOLN_PATH + j * DIMENSIONS * LENGTH_OF_SOLN_PATH + DIMENSIONS * i + 1] = x_path[j][i][1];
			control_solutions[idx * num_of_goals * LENGTH_OF_SOLN_PATH + j * LENGTH_OF_SOLN_PATH + i] = u_path[j][i];
			if (not_found[j] == 0) {
				if (i == LENGTH_OF_SOLN_PATH - 2) {
					path_solutions[idx * DIMENSIONS * num_of_goals * LENGTH_OF_SOLN_PATH + j * DIMENSIONS * LENGTH_OF_SOLN_PATH + DIMENSIONS * i] = start_state[0];
					path_solutions[idx * DIMENSIONS * num_of_goals * LENGTH_OF_SOLN_PATH + j * DIMENSIONS * LENGTH_OF_SOLN_PATH + DIMENSIONS * i + 1] = start_state[1];
				} else if (i == LENGTH_OF_SOLN_PATH - 1) {
					path_solutions[idx * DIMENSIONS * num_of_goals * LENGTH_OF_SOLN_PATH + j * DIMENSIONS * LENGTH_OF_SOLN_PATH + DIMENSIONS * i] = end_state[j][0];
					path_solutions[idx * DIMENSIONS * num_of_goals * LENGTH_OF_SOLN_PATH + j * DIMENSIONS * LENGTH_OF_SOLN_PATH + DIMENSIONS * i + 1] = end_state[j][1];
				}
			}
		}
	}
	//*/


	/*
	 int i;
	 for (i = 0; i < NUM_RESULTS_PER_THREAD; i++)
	 result[idx * NUM_RESULTS_PER_THREAD + i] = start_state[i];
	 //*/
	/*
	 result[idx * NUM_RESULTS_PER_THREAD + 0] = start_state[0];
	 result[idx * NUM_RESULTS_PER_THREAD + 1] = start_state[1];
	 //*/

}



////////////////////////////////////////////
// HELPER FUNCTIONS
////////////////////////////////////////////
/*
 * computes the Euclidian distances squared from point A to every point in array B
 */
__device__ void euclidianDistSquare(double* A, double B[][2], int lengthOfB,
		double* listOfDistSq) {
	int i;
	for (i = 0; i < lengthOfB; i++)
		listOfDistSq[i] = pow((B[i][0] - A[0]), 2) + pow((B[i][1] - A[1]), 2);
}

/*
 * finds the index of the minimum in an array
 */
__device__ int findMin(double array[], int lengthOfArray) {
	int minIndex = 0;

	int i;
	for (i = 0; i < lengthOfArray; i++) {
		if (array[i] < array[minIndex])
			minIndex = i;
	}

	return minIndex;
}

/*
 * Computes x_dot of the pendulum, given x and a control input u
 */
__device__ void pendulumDynamics(double* x, double u, double* next_state) {
	// pendulum parameters
	int m = 1;                  // mass
	int l = 1;                  // length of pendulum link
	int I = m * l * l;              // moment of inertia
	double g = 9.8;              // acceleration due to gravity
	double b = 0.1;              // damping factor

	next_state[0] = x[1];
	next_state[1] = (u - m * g * l * sin((M_PI / 2) - x[0]) - b * x[1]) / I;
}
