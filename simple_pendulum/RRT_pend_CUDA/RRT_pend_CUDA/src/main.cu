#include "hip/hip_runtime.h"
////////////////////////////////////////////
// INCLUDES
////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#include <vector>
#include "floyd_warshall_algo.cuh"
#include "RRT.cuh"
using namespace std;

////////////////////////////////////////////
// MAIN
////////////////////////////////////////////
int main(void) {

	// Allocate memory for Random Number Generator kernel
	hiprandState *device_state;
	hipMalloc(&device_state, NUM_THREADS * NUM_BLOCKS * sizeof(hiprandState)); // allocate device memory to store RNG states

	// Initialize variables to store results on PC (host) and GPU (device) (i.e.: adjacency matrix, paths & control actions)
	int *device_adjacency_matrix, *host_adjacency_matrix;
	double *device_path_solns, *host_path_solns;
	double *device_control_solns, *host_control_solns;

	// Used to get the start state for each thread
	double *dev_start_state_0, *host_start_state_0;
	host_start_state_0 = (double *) malloc(2 * 2016 * sizeof(double));
	hipMalloc(&dev_start_state_0, 2 * 2016 * sizeof(double));

	// Allocate host and device memory to store adjacency matrix
	host_adjacency_matrix = (int *) malloc(
			NUM_RESULTS_PER_THREAD * NUM_THREADS * NUM_BLOCKS * sizeof(int));// allocate host memory to store adjacency matrix
	hipMalloc(&device_adjacency_matrix,
			NUM_RESULTS_PER_THREAD * NUM_THREADS * NUM_BLOCKS * sizeof(int));// allocate device memory to store adjacency matrix
	init_adj_matrix_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(device_adjacency_matrix);	// initialize adjacency matrix in parallel

	// Allocate host and device memory to store path solution for each tree
	host_path_solns = (double *) malloc(
			NUM_RESULTS_PER_THREAD_2 * NUM_THREADS * NUM_BLOCKS * sizeof(double));// allocate host memory to store path solutions
	hipMalloc(&device_path_solns,
			NUM_RESULTS_PER_THREAD_2 * NUM_THREADS * NUM_BLOCKS * sizeof(double));// allocate device memory to store path solutions
	hipMemset(device_path_solns, 0,
			NUM_RESULTS_PER_THREAD_2 * NUM_THREADS * NUM_BLOCKS * sizeof(double));// initialize device results array to 0

	// Allocate host and device memory to store control solutions for each tree
	host_control_solns = (double *) malloc(
			NUM_OF_GOAL_STATES* LENGTH_OF_SOLN_PATH * NUM_THREADS * NUM_BLOCKS * sizeof(double));// allocate host memory to store control solutions
	hipMalloc(&device_control_solns,
			NUM_OF_GOAL_STATES* LENGTH_OF_SOLN_PATH * NUM_THREADS * NUM_BLOCKS * sizeof(double));// allocate device memory to store control solutions
	hipMemset(device_control_solns, 0,
			NUM_OF_GOAL_STATES* LENGTH_OF_SOLN_PATH * NUM_THREADS * NUM_BLOCKS * sizeof(double));// initialize device results array to 0

	// Initialize RNG in parallel
	RNG_setup_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(device_state);

	// Run parallel RRT algorithm
	RRT_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(device_state, device_adjacency_matrix,
			device_path_solns, device_control_solns, dev_start_state_0);

	// Copy results from GPU (device) to PC (host)
	hipMemcpy(host_adjacency_matrix, device_adjacency_matrix,
			NUM_RESULTS_PER_THREAD * NUM_THREADS * NUM_BLOCKS * sizeof(int),	// copy adjacency matrix from device to host
			hipMemcpyDeviceToHost);

	hipMemcpy(host_path_solns, device_path_solns,
			NUM_RESULTS_PER_THREAD_2 * NUM_THREADS * NUM_BLOCKS * sizeof(double),// copy path solutions from device to host
			hipMemcpyDeviceToHost);

	hipMemcpy(host_control_solns, device_control_solns,
			NUM_OF_GOAL_STATES* LENGTH_OF_SOLN_PATH * NUM_THREADS * NUM_BLOCKS * sizeof(double),// copy control solutions from device to host
			hipMemcpyDeviceToHost);

	// copy start state of each thread and output to console
	hipMemcpy(host_start_state_0, dev_start_state_0, 2 * 2016 * sizeof(double), hipMemcpyDeviceToHost);
	for(int i=0; i < 2016; i++)
			printf("idx: %d State:%f, %f\n",i,host_start_state_0[2*i],host_start_state_0[2*i+1]);


	// Initialize variables used to obtain shortest path solution using Floyd-Warshall algorithm
	int *device_roots_idx_on_path=(int *)malloc(NUM_RESULTS_PER_THREAD * NUM_THREADS * NUM_BLOCKS * sizeof(int));
	int *root_indices = (int *)calloc(29, sizeof(int));

	for(int i=0;i<RANDOM_GSIZE*RANDOM_GSIZE;i++){
		device_roots_idx_on_path[i]=-1;//set to all negative ones for use in path construction
	}

	// Call host function which will copy all info to device and run Floyd-Warshall CUDA kernels
	_GPU_Floyd(host_adjacency_matrix,device_roots_idx_on_path,RANDOM_GSIZE);

	// Find out exact step-by-step shortest paths between vertices(if such a path exists)
	_get_full_paths(host_adjacency_matrix,device_roots_idx_on_path,RANDOM_GSIZE, root_indices);

	/* output root indexes on solution path
	for(int i=0; i < 30;i++){
		printf("%d\n",root_indices[i]);
	}
	//*/

	/* Root indices on solution path (using this since I already know the solution from previous run)
	root_indices[0] = 975;
	root_indices[1] = 1017;
	root_indices[2] = 1059;
	root_indices[3] = 1101;
	root_indices[4] = 1143;
	root_indices[5] = 1185;
	root_indices[6] = 1228;
	root_indices[7] = 1271;
	root_indices[8] = 1314;
	root_indices[9] = 1315;
	root_indices[10] = 1316;
	root_indices[11] = 1317;
	root_indices[12] = 1318;
	root_indices[13] = 1319;
	root_indices[14] = 1320;
	root_indices[15] = 1321;
	root_indices[16] = 1322;
	root_indices[17] = 1323;
	root_indices[18] = 1282;
	root_indices[19] = 1241;
	root_indices[20] = 1200;
	root_indices[21] = 1159;
	root_indices[22] = 1118;
	root_indices[23] = 1077;
	root_indices[24] = 1036;
	root_indices[25] = 995;
	root_indices[26] = 1038;
	root_indices[27] = 1039;
	root_indices[28] = 997;
	//*/
	//*
	root_indices[0] = 1018;
	root_indices[1] = 1060;
	root_indices[2] = 1102;
	root_indices[3] = 1144;
	root_indices[4] = 1186;
	root_indices[5] = 1228;
	root_indices[6] = 1270;
	root_indices[7] = 1313;
	root_indices[8] = 1356;
	root_indices[9] = 1357;
	root_indices[10] = 1358;
	root_indices[11] = 1359;
	root_indices[12] = 1360;
	root_indices[13] = 1361;
	root_indices[14] = 1362;
	root_indices[15] = 1363;
	root_indices[16] = 1364;
	root_indices[17] = 1323;
	root_indices[18] = 1282;
	root_indices[19] = 1241;
	root_indices[20] = 1200;
	root_indices[21] = 1159;
	root_indices[22] = 1118;
	root_indices[23] = 1077;
	root_indices[24] = 1036;
	root_indices[25] = 1037;
	root_indices[26] = 1038;
	root_indices[27] = 1039;
		//*/


	//* Construct full solution path
	std::vector< std::pair<std::pair<double,double>, double> > solution_path;
	int root_index;
	int next_root_index;
	int start_of_path, start_of_next_path;
	double root_x, root_y, goal_x, goal_y;
	for(int i=0; i < 28; i++){
		root_index = root_indices[i];
		next_root_index = root_indices[i+1];
		for(int goal_index=0; goal_index < 7; goal_index++){
			start_of_path = 2*NUM_OF_GOAL_STATES*LENGTH_OF_SOLN_PATH*root_index + goal_index*2*LENGTH_OF_SOLN_PATH;
			start_of_next_path = 2*NUM_OF_GOAL_STATES*LENGTH_OF_SOLN_PATH*next_root_index + goal_index*2*LENGTH_OF_SOLN_PATH;
			root_x = host_path_solns[start_of_next_path-4+(2*LENGTH_OF_SOLN_PATH)];
			root_y = host_path_solns[start_of_next_path-3+(2*LENGTH_OF_SOLN_PATH)];
			goal_x = host_path_solns[start_of_path-2+(2*LENGTH_OF_SOLN_PATH)];
			goal_y = host_path_solns[start_of_path-1+(2*LENGTH_OF_SOLN_PATH)];

			if((root_x+root_y != 0.000000) && (fabs(root_x-goal_x) < 0.000001) && (fabs(root_y-goal_y) < 0.000001)){

				//solution_path.push_back(std::make_pair(host_path_solns[start_of_path-4+40], host_path_solns[start_of_path-3+40]));
				for(int i=0; i < LENGTH_OF_SOLN_PATH-2; i++){
					if((host_path_solns[start_of_path+i] == 0.000000) && (host_path_solns[start_of_path+i+1] == 0.000000))
						break;

					//printf("%f\n", host_path_solns[start_of_path+i]);
					solution_path.push_back(std::make_pair(std::make_pair(host_path_solns[start_of_path+(2*i)], host_path_solns[start_of_path+(2*i+1)]), host_control_solns[(start_of_path/2)+i] ));
				}
			}
		}
	}
	//*/

	/* output full solution path
	for(int i=0; i < solution_path.size(); ++i){
		if(solution_path[i].first.first + solution_path[i].first.second != 0)
			std::cout << solution_path[i].first.first << " " << solution_path[i].first.second << " " << solution_path[i].second << std::endl;
	}
	//*/

	//*
	// Output results to files
	FILE *adjacency_matrix_file = fopen("adjacency_matrix.txt", "w");
	FILE *path_solutions_file = fopen("path_solutions.txt", "w");
	FILE *control_solutions_file = fopen("control_solutions.txt", "w");
	FILE *pendulum_estimates_file = fopen("pendulum_estimates.txt", "w");
	FILE *state_index_file = fopen("state_index.txt", "w");

	for (int i = 0; i < NUM_THREADS * NUM_BLOCKS; i++) {
		for (int j = 0; j < NUM_RESULTS_PER_THREAD; j++) {
			fprintf(adjacency_matrix_file, "%d ", host_adjacency_matrix[i * NUM_THREADS * NUM_BLOCKS + j]);
		}
		fprintf(adjacency_matrix_file, "\n");
	}

	for (int i = 0; i < NUM_RESULTS_PER_THREAD_2 * NUM_THREADS * NUM_BLOCKS; i++)
		fprintf(path_solutions_file, "%f,\n", host_path_solns[i]);

	for (int i = 0; i < NUM_OF_GOAL_STATES* LENGTH_OF_SOLN_PATH * NUM_THREADS * NUM_BLOCKS; i++)
		fprintf(control_solutions_file, "%f,\n", host_control_solns[i]);

	//*
	for(int i=0; i < solution_path.size(); ++i){
		if(solution_path[i].first.first + solution_path[i].first.second != 0)
			fprintf(pendulum_estimates_file, "%f %f %f\n", solution_path[i].first.first, solution_path[i].first.second, solution_path[i].second);
	}
	//*/

	for(int i=0; i < NUM_RESULTS_PER_THREAD; i++)
			fprintf(state_index_file, "idx: %d State:%f, %f\n",i,host_start_state_0[2*i],host_start_state_0[2*i+1]);

	fclose(adjacency_matrix_file);
	fclose(path_solutions_file);
	fclose(control_solutions_file);
	fclose(pendulum_estimates_file);
	fclose(state_index_file);
	//*/

	// Free memory
	free(host_path_solns);
	free(host_control_solns);
	free(host_adjacency_matrix);
	free(device_roots_idx_on_path);
	free(host_start_state_0);
	hipFree(device_path_solns);
	hipFree(device_control_solns);
	hipFree(device_adjacency_matrix);
	hipFree(device_state);
	hipFree(dev_start_state_0);

	return 0;
}
